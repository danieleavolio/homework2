#include "hip/hip_runtime.h"
#include <stdint.h>
#include <iostream>
#include <string>
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define CHANNEL_NUM 3

#include "stb_image.h"
#include "stb_image_write.h"

using namespace std;

void checkCudaError()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
}

__global__ void kernelFunction(unsigned char *Pout, unsigned char *Pin, int width,
                            int height, int channels)
{
    // Pout and Pin point to 1 dimensional array
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    int size = width * height * channels;
    // for (int i = 0; i < size; i+=channels){
    // int media = (Pin[i] + Pin[i + 1] + Pin[i+2]) / 3;
    // Pout[i] = Pout[i + 1] = Pout[i + 2] = media;
    // if (channels == 4) Pout[i+3] = Pin[i+3];
    //}
}


int main(int argc, char **argv)
{
    int width, height, channel;

    unsigned char *bw_image;
    unsigned char *rgb_image;
    unsigned char *cpu_rgb_image;

    const char *input_name = argv[1];
    const char *output_name = argv[2];

    if (argc != 3)
    {
        cout << "There was an error. Insert the correct number of arguments" << endl;
        exit(EXIT_FAILURE);
    }
    cpu_rgb_image = stbi_load(input_name, &width, &height, &channel, 0);
    int size = width * height * sizeof(unsigned char) * channel;
    cout << "Alloco la memoria necessaria \n";
    hipMalloc(&rgb_image, size);
    hipMallocManaged(&bw_image, size);
    int block_size = 32;
    int number_of_blocks = ceil((width * height * channel) / block_size);
    cout << "Copio sulla GPU \n";
    hipMemcpy(rgb_image, cpu_rgb_image, size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    cout << "Chiamo la funzione kernel \n";
    cout << width << " " << height << " " << channel << endl;
    kernelFunction<<<number_of_blocks, block_size>>>(bw_image, rgb_image, width, height, channel);
    hipDeviceSynchronize();
    cout << "Controllo gli errori \n";
    checkCudaError();
    cout << "Scrivo l'immagine \n";
    stbi_write_jpg(output_name, width, height, channel, bw_image, 100);
    cout << "Libero tutto \n";
    hipFree(bw_image);
    hipFree(rgb_image);
    hipFree(cpu_rgb_image);

    delete bw_image;
    delete cpu_rgb_image;
    delete rgb_image;

    return 0;
}