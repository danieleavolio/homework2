#include "hip/hip_runtime.h"
#include <stdint.h>
#include <iostream>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define CHANNEL_NUM 3

#include "stb_image.h"
#include "stb_image_write.h"

using namespace std;

__global__ void colorToGrey(unsigned char *Pout, unsigned char *Pin, int width,
                            int height, int channels)
{
    // Pout and Pin point to 1 dimensional array
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    int size = width * height * channels;
    // for (int i = 0; i < size; i+=channels){
    int media = (Pin[i] + Pin[i + 1] + Pin[1 + 2]) / 3;
    Pout[i] = Pout[i + 1] = Pout[i + 2] = media;
    if (channels == 4) Pout[i+3] = Pin[i+3];
    //}
}

void serialeBw(unsigned char *Pout, unsigned char *Pin, int width, int height, int channels)
{

    //int i = blockDim.x * blockIdx.x + threadIdx.x;
//
    //int size = width * height * channels;
    //// for (int i = 0; i < size; i+=channels){
    //int media = (Pin[i] + Pin[i + 1] + Pin[1 + 2]) / 3;
    //Pout[i] = Pout[i + 1] = Pout[i + 2] = media;
    ////}
    //stbi_write_png("image_bw.png", width, height, channels, Pout, width * channels);
}

struct Pixel
{
    unsigned char r, g, b, a;
};

void checkCudaError()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
}

void convertImageToGrayCPU(unsigned char *rgb_image, int width, int height)
{
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            // for every pixel
            Pixel *ptrPixel = (Pixel *)&rgb_image[y * width * 3 + 3 * x];
            unsigned char pixelValue = (unsigned char)ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f;
            ptrPixel->r = pixelValue;
            ptrPixel->g = pixelValue;
            ptrPixel->b = pixelValue;
            ptrPixel->a = 255;
        }
    }
}

int main()
{
    int width, height, channel;

    unsigned char *bw_image;
    unsigned char *rgb_image;
    unsigned char *cpu_rgb_image;
    cpu_rgb_image = stbi_load("godrick.jpg", &width, &height, &channel, 0);
    int size = width * height * sizeof(unsigned char) * channel;
    cout << "Alloco la memoria necessaria \n";
    hipMalloc(&rgb_image, size);
    hipMallocManaged(&bw_image, size);
    int block_size = 32;
    int number_of_blocks = ceil((width * height * channel) / block_size);
    cout << "Copio sulla GPU \n";
    hipMemcpy(rgb_image, cpu_rgb_image, size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    cout << "Chiamo la funzione kernel \n";
    cout << width << " " << height << " " << channel << endl;
    colorToGrey<<<number_of_blocks, block_size>>>(bw_image, rgb_image, width, height, channel);
    hipDeviceSynchronize();
    cout << "Controllo gli errori \n";
    cout << "Copio sulla CPU\n";
    checkCudaError();
    // serialeBw(bw_image, cpu_rgb_image, width, height, channel);
    stbi_write_png("image_bw.png", width, height, channel, bw_image, width * channel);
    cout << "Libero tutto \n";
    hipFree(bw_image);
    hipFree(rgb_image);
    hipFree(cpu_rgb_image);

    return 0;
}